#include "hip/hip_runtime.h"
#pragma once
#include "CUDAModel.cuh"
#include <thrust\reduce.h>
#include <thrust\scan.h>
#include <thrust\transform.h>
#include <thrust\pair.h>
#include <thrust\reduce.h>
#include <thrust\sort.h>
#include <thrust\unique.h>
#include <thrust\functional.h>
#include <thrust\for_each.h>
#include <thrust\system\cuda\execution_policy.h>

extern "C" bool DataTransfer(XMLModel *model);
extern "C" bool BuildArcsModel(XMLModel *model);
extern "C" int AC4GpuPlusInitialization();
extern "C" int AC4GpuPropagation();

#define CSCOUNT 3
typedef thrust::tuple<int, int> Node;
typedef thrust::device_vector<Node> Nodes;
typedef thrust::device_vector<int>::iterator   IntIterator;
typedef thrust::tuple<IntIterator, IntIterator, IntIterator, IntIterator> D_ArcIterTuple;
typedef thrust::tuple<IntIterator, IntIterator, IntIterator, IntIterator, IntIterator, IntIterator> D_ArcSorcIterTuple;
typedef thrust::tuple<IntIterator, IntIterator, IntIterator> D_CounterIterTuple;
typedef thrust::tuple<IntIterator, IntIterator> D_NodesIterTuple;
typedef thrust::zip_iterator<D_ArcIterTuple> D_ArcTupleIter;
typedef thrust::zip_iterator<D_CounterIterTuple> D_CounterIter;
typedef thrust::zip_iterator<D_NodesIterTuple> D_NodesIter;
typedef thrust::zip_iterator<D_ArcSorcIterTuple> D_ArcSorcIter;
typedef thrust::pair<D_CounterIter, IntIterator> Counter;
const static int MAXTHREADSPERBLOCK = 1024;
const static int MaxThreads = 10240000;
int dbsum;
int stream_size;
bool propagationEnable = true;
thrust::device_vector<int> d_vars_size;
thrust::host_vector<int> h_vars_size;
//��ƫ����
thrust::device_vector<int> d_node_global;
thrust::device_vector<int> d_nodes_set;
//thrust::device_vector<int> d_segment_indexes;

//counterƫ����
//thrust::device_vector<int> d_local_counter;
//counter����
int counter_sum;
int nodes_sum;
__device__ bool d_is_sat;
__device__ bool d_conti;

struct D_Arcs
{
	thrust::device_vector<int> d_vars0;
	thrust::device_vector<int> d_vals0;
	thrust::device_vector<int> d_vars1;
	thrust::device_vector<int> d_vals1;
	thrust::device_vector<int> d_sorcs;
	thrust::device_vector<int> d_cmaps;

	D_Arcs()
	{
	}

	D_Arcs(size_t len)
	{
		resize(len);
	}

	void operator()(size_t len)
	{
		resize(len);
	}

	void resize(size_t len)
	{
		d_vars0.resize(len);
		d_vals0.resize(len);
		d_vars1.resize(len);
		d_vals1.resize(len);
		d_sorcs.resize(len);
		d_cmaps.resize(len);
	}

	//void erase()

}d_arcs, d_arcs2;

struct D_Vars_Size
{
	thrust::device_vector<int> vars;
	thrust::device_vector<int> sizes;
	D_Vars_Size(){}
	D_Vars_Size(size_t len)
	{
		resize(len);
	}

	void resize(size_t len)
	{
		vars.resize(len);
		sizes.resize(len);
	}
};

struct H_Arcs
{
	thrust::host_vector<int> h_vars0;
	thrust::host_vector<int> h_vals0;
	thrust::host_vector<int> h_vars1;
	thrust::host_vector<int> h_vals1;
	thrust::host_vector<int> h_sorcs;
	thrust::host_vector<int> h_cmaps;

	void operator= (D_Arcs das)
	{
		h_vars0 = das.d_vars0;
		h_vals0 = das.d_vals0;
		h_vars1 = das.d_vars1;
		h_vals1 = das.d_vals1;
		h_sorcs = das.d_sorcs;
		h_cmaps = das.d_cmaps;
	}
}h_arcs;

struct D_Node
{
	thrust::device_vector<int> vars;
	thrust::device_vector<int> vals;
	D_Node()
	{
	}

	D_Node(size_t len)
	{
		vars.resize(len);
		vals.resize(len);
	}

	void operator()(size_t len)
	{
		resize(len);
	}

	void resize(size_t len)
	{
		vars.resize(len);
		vals.resize(len);
	}
}d_nodes;

struct 	D_SegmentIndex
{
	thrust::device_vector<int> start;
	thrust::device_vector<int> end;

	D_SegmentIndex()
	{
	}

	D_SegmentIndex(size_t len)
	{
		resize(len);
	}

	D_SegmentIndex(size_t len, int init_value)
	{
		resize(len, init_value);
	}

	void resize(size_t len)
	{
		start.resize(len);
		end.resize(len);
	}

	void resize(size_t len, int init_value)
	{
		start.resize(len, init_value);
		end.resize(len, init_value);
	}
}d_segidx;

__global__ void ComputeLocalOffset(int *d_offset, int *d_local_counter, XMLConstraint *d_c, XMLDomain *d_d, XMLVariable *d_v)
{
	int i = threadIdx.x;
	int x = d_c[i].scope.x;
	int y = d_c[i].scope.y;
	int x_dm_id = d_v[x].dm_id;
	int y_dm_id = d_v[y].dm_id;
	int x_dm_size = d_d[x_dm_id].size;
	int y_dm_size = d_d[y_dm_id].size;
	int local_offset = x_dm_size*y_dm_size;
	d_offset[i] = 2 * local_offset;
	d_local_counter[i] = x_dm_size + y_dm_size;
}

__global__ void GenerateVars_size(int *vars_size, XMLVariable *vars, XMLDomain *dms)
{
	int i = threadIdx.x;
	int dm_id = vars[i].dm_id;
	int dm_size = dms[dm_id].size;
	vars_size[i] = dm_size;
}

__global__ void GenerateNodes(int *node_var, int *node_val, int *offset, int var_id)
{
	int i = threadIdx.x;
	int idx = offset[var_id] + i;
	node_var[idx] = var_id;
	node_val[idx] = i;
}

__global__ void GenerateNodesLaunch(int *node_var, int *node_val, int *offset, XMLVariable *vars, XMLDomain *dms)
{
	int i = threadIdx.x;
	int dm_id = vars[i].dm_id;
	int dm_size = dms[dm_id].size;
	GenerateNodes << <1, dm_size >> >(node_var, node_val, offset, i);
}

__global__
void BuildArc(
int *d_vars0,
int *d_vals0,
int *d_vars1,
int *d_vals1,
int *d_sorc,
int *d_cmap,
int d_offset,
int d_global_offset,
int d_global_counter,
XMLDomain dm_0,
XMLDomain dm_1,
XMLVariable var_0,
XMLVariable var_1,
XMLConstraint cst, int sorc
)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	int val0 = dm_0.values[x];
	int val1 = dm_1.values[y];
	int var0 = var_0.id;
	int var1 = var_1.id;
	int local_arc0 = blockDim.x*x + y;
	int local_arc1 = gridDim.x*y + x;
	int global_arc0 = local_arc0 + d_global_offset;
	int global_arc1 = local_arc1 + d_offset / 2 + d_global_offset;
	int global_counter0 = d_global_counter + x;
	int global_counter1 = d_global_counter + gridDim.x + y;
	//printf("%d %d %d %d,%d %d %d\n", var0, val0, var1, val1, sorc, global_counter0, global_counter1);

	d_vars0[global_arc0] = var0;
	d_vals0[global_arc0] = val0;
	d_vars1[global_arc0] = var1;
	d_vals1[global_arc0] = val1;
	d_vars0[global_arc1] = var1;
	d_vals0[global_arc1] = val1;
	d_vars1[global_arc1] = var0;
	d_vals1[global_arc1] = val0;
	d_sorc[global_arc0] = sorc;
	d_sorc[global_arc1] = sorc;
	d_cmap[global_arc0] = global_counter0;
	d_cmap[global_arc1] = global_counter1;

	return;
}

__global__
void ModifyTuple(
int* d_sorc,
int d_offset,
int d_global_offset,
XMLDomain dm_0,
XMLDomain dm_1,
XMLRelation rel,
int semantices,
int size
)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < size)
	{
		bituple bt = rel.tuples[i];
		int var0_size = dm_0.size;
		int var1_size = dm_1.size;
		int sorc = semantices;
		int local_sorc0 = bt.x*var1_size + bt.y;
		int local_sorc1 = bt.y*var0_size + bt.x;
		int global_sorc0 = local_sorc0 + d_global_offset;
		int global_sorc1 = local_sorc1 + d_offset / 2 + d_global_offset;
		d_sorc[global_sorc0] = sorc;
		d_sorc[global_sorc1] = sorc;
	}
}

__global__
void BuildArcsLaunch(
int *d_vars0,
int *d_vals0,
int *d_vars1,
int *d_vals1,
int *d_sorc,
int *d_cmap,
int *d_offset,
int *d_global_offset,
int *d_global_counter,
XMLDomain *dms,
XMLVariable *vars,
XMLRelation *rels,
XMLConstraint *csts
)
{
	int mtpb = 1024;
	int i = threadIdx.x;
	XMLConstraint constraint = csts[i];
	int x = constraint.scope.x;
	int y = constraint.scope.y;
	XMLRelation relation = rels[constraint.re_id];
	int r_size = relation.size;
	XMLVariable var0 = vars[x];
	XMLVariable var1 = vars[y];
	int x_dm_id = var0.dm_id;
	int y_dm_id = var1.dm_id;
	XMLDomain dm_0 = dms[x_dm_id];
	XMLDomain dm_1 = dms[y_dm_id];
	int x_size = dm_0.size;
	int y_size = dm_1.size;
	int semantices = relation.semantices;
	int sorc = !semantices;
	int block_size = r_size / (mtpb)+!(!(r_size % (mtpb)));
	BuildArc << <x_size, y_size >> >(d_vars0, d_vals0, d_vars1, d_vals1, d_sorc, d_cmap, d_offset[i], d_global_offset[i], d_global_counter[i], dm_0, dm_1, var0, var1, constraint, sorc);
	__syncthreads();
	ModifyTuple << <block_size, mtpb >> >(d_sorc, d_offset[i], d_global_offset[i], dm_0, dm_1, relation, semantices, r_size);
}

__global__ void Vars_Resize(int *var, int *del, int*vars)
{
	int i = threadIdx.x;
	int del_var_id = var[i];
	int del_now = del[i];
	int size = vars[del_var_id];
	int new_size = size - del_now;
	vars[del_var_id] = new_size;
}

__global__ void show(int *a)
{
	int i = threadIdx.x;
	printf("%d\n", a[i]);
}

__global__ void CompareVar_Size(int *old_var_size, int *tmp_var_size, int size)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	d_is_sat = true;
	d_conti = false;

	if (idx < size)
	{
		int old = old_var_size[idx];
		int tmp = tmp_var_size[idx];
		if (old != tmp)
		{
			d_conti = true;
		}
		if (tmp == 0)
		{
			d_is_sat = false;
		}
	}
}

struct TernaryPredicate
{
	template<typename Tuple>
	__host__ __device__ bool operator()(const Tuple& a, const Tuple& b)
	{
		return(
			(thrust::get<0>(a) == thrust::get<0>(b)) &&
			(thrust::get<1>(a) == thrust::get<1>(b)) &&
			(thrust::get<2>(a) == (thrust::get<2>(b)))
			);
	}
};

struct is_conflict
{
	template<typename Tuple>
	//__host__ __device__ bool operator()(const thrust::tuple<const int&, const int&, const int&, const int&, const int&, const int&> &a)
	__host__ __device__ bool operator()(const Tuple& a)
	{
		return (!(thrust::get<4>(a)));
	}
};

struct is_zero
{
	int *nodes;
	int *offset;
	int len;
	is_zero(int *nodes, int *offset, int len) :len(len), nodes(nodes), offset(offset){}
	is_zero(){}
	__host__ __device__
		bool operator()(const int s_val) const {
			return (!s_val);
		}
};

struct ModifyNodes
{
	int *nodes;
	int *node_offset;
	ModifyNodes(int *nodes, int *node_offset) :nodes(nodes), node_offset(node_offset){}

	template<typename Tuple>
	__host__ __device__	void operator()(const Tuple &t)
	{
		int var = thrust::get<0>(t);
		int val = thrust::get<1>(t);
		int sorc = thrust::get<2>(t);
		if (sorc == 0)
		{
			//thrust::get<2>(t) = 0;
			//printf("-1\n");
			int node_idx = node_offset[var] + val;
			nodes[node_idx] = 0;
		}
	}
};

//struct ModifyNodesProp
//{
//	int *nodes;
//	int *node_offset;
//	int *d_vars_size_tmp;
//	ModifyNodesProp(int *nodes, int *node_offset, int *d_vars_size_tmp) :nodes(nodes), node_offset(node_offset), d_vars_size_tmp(d_vars_size_tmp){}
//
//	template<typename Tuple>
//	__device__	void operator()(const Tuple &t)
//	{
//		int var = thrust::get<0>(t);
//		int val = thrust::get<1>(t);
//		int sorc = thrust::get<2>(t);
//		if (sorc == -1)
//		{
//			thrust::get<2>(t) = 0;
//			int node_idx = node_offset[var] + val;
//			int e = nodes[node_idx];
//			if (e == 1)
//			{
//				nodes[node_idx] = 0;
//				atomicAdd(&d_vars_size_tmp[var], -1);
//				if (d_vars_size_tmp[var] < 0)
//				{
//					d_vars_size_tmp[var] = 0;
//				}
//			}
//		}
//	}
//};

struct ModifyNodesProp
{
	int *nodes;
	int *node_offset;
	int *d_vars_size_tmp;
	ModifyNodesProp(int *nodes, int *node_offset, int *d_vars_size_tmp) :nodes(nodes), node_offset(node_offset), d_vars_size_tmp(d_vars_size_tmp){}

	template<typename Tuple>
	__device__	void operator()(const Tuple &t)
	{
		int var = thrust::get<0>(t);
		int val = thrust::get<1>(t);
		int sorc = thrust::get<2>(t);
		if (sorc == -1)
		{
			thrust::get<2>(t) = 0;
			int node_idx = node_offset[var] + val;
			int e = nodes[node_idx];
			atomicAdd(&d_vars_size_tmp[var], 0 - nodes[node_idx]);
			nodes[node_idx] = 0;
			if (d_vars_size_tmp[var] < 0)
			{
				d_vars_size_tmp[var] = 0;
			}
		}
	}
};

struct ModifyArcs
{
	int *nodes;
	int *node_offset;
	int *counter_value;
	ModifyArcs(int *nodes, int *node_offset, int *counter_value) :nodes(nodes), node_offset(node_offset), counter_value(counter_value){}

	template<typename Tuple>
	__device__	void operator()(const Tuple &t)
	{
		int var0 = thrust::get<0>(t);
		int val0 = thrust::get<1>(t);
		int var1 = thrust::get<2>(t);
		int val1 = thrust::get<3>(t);
		int sorc = thrust::get<4>(t);
		int cmap = thrust::get<5>(t);

		int s0 = nodes[node_offset[var0] + val0];
		int s1 = nodes[node_offset[var1] + val1];

		if (sorc == 1 && s0 == 0)
		{
			thrust::get<4>(t) = 0;
			return;
		}

		//bool del_node = !(nodes[node_offset[var0] + val0] && nodes[node_offset[var1] + val1]);

		if (sorc == 1 && s1 == 0)
		{
			thrust::get<4>(t) = 0;
			atomicAdd(&counter_value[cmap], -1);

			if (counter_value[cmap] == 0)
			{
				counter_value[cmap] = -1;
				//printf("counter = %d\n", counter_value[cmap]);
				//printf("00\n");
			}
			return;
		}
	}
};

struct DeleteNodes
{
	int *nodes;
	int *offset;
	int len;

	DeleteNodes(int *nodes, int *offset, int len) :len(len), nodes(nodes), offset(offset){}

	template <typename Tuple>
	__host__ __device__ void operator()(const Tuple &t)
	{
		int var = thrust::get<0>(t);
		int val = thrust::get<1>(t);
		nodes[offset[var] + val] = 0;
	}
};

struct Is_Deleted
{
	int *nodes;
	int *offset;
	int len;
	Is_Deleted(int *nodes, int *offset, int len) : nodes(nodes), offset(offset), len(len) {}

	template <typename Tuple>
	__host__ __device__ bool operator()(const Tuple &t)
	{
		int var = thrust::get<0>(t);
		int val = thrust::get<1>(t);
		int global_offset = offset[var] + val;
		bool deleted = nodes[global_offset];
		return !deleted;
	}
};

struct Build_Segment_index
{
	int *seg;
	int *g_offset;
	int *start;
	int *end;
	int arcdim;
	Build_Segment_index(int *seg, int *g_offset, int arcdim) :seg(seg), g_offset(g_offset), arcdim(arcdim){}
	Build_Segment_index(int *start, int *end, int *g_offset, int arcdim) :start(start), end(end), g_offset(g_offset), arcdim(arcdim){}
	__host__ __device__ void operator()(const int &idx)
	{
		int offset = g_offset[idx];
		int offset_pre = g_offset[idx - 1];
		int segment = offset / arcdim;
		int segment_pre = offset_pre / arcdim;

		if (segment != segment_pre)
		{
			start[segment] = offset_pre;

			if ((segment - 1) >= 0)
			{
				end[segment - 1] = offset_pre - 1;
			}
		}
	}
};

XMLDomain *h_dms;
XMLDomain *d_dms;
XMLVariable *d_vs;
XMLRelation *h_rs;
XMLRelation *d_rs;
XMLConstraint *d_cs;

extern "C" bool DataTransfer(XMLModel *model)
{
#pragma region ����������
	int ds_size = model->ds_size;
	int ds_len = ds_size *sizeof(XMLDomain);
	int d_size;
	h_dms = new XMLDomain[ds_size];
	memcpy(h_dms, model->domains, ds_len);

	for (size_t i = 0; i < ds_size; ++i)
	{
		d_size = model->domains[i].size;
		hipMalloc(&(h_dms[i].values), d_size*sizeof(int));
		hipMemcpy(h_dms[i].values, model->domains[i].values, d_size*sizeof(int), hipMemcpyHostToDevice);
	}

	hipMalloc((void**)&d_dms, ds_len);
	hipMemcpy(d_dms, h_dms, ds_len, hipMemcpyHostToDevice);
	//ShowDomains << <ds_size, d_size >> >(d_dms);
#pragma endregion

#pragma region ������������
	int vs_size = model->vs_size;
	int vs_len = vs_size*sizeof(XMLVariable);
	//int v_size;
	hipMalloc((void **)&d_vs, vs_len);
	hipMemcpy(d_vs, model->variables, vs_len, hipMemcpyHostToDevice);
	//ShowVariables << <1, vs_size >> >(d_vs);
#pragma endregion

#pragma region ������ϵ����
	int rs_size = model->rs_size;
	int rs_len = rs_size*sizeof(XMLRelation);
	int r_size;
	int r_maxsize = 0;
	int r_len;
	h_rs = new XMLRelation[rs_size];
	memcpy(h_rs, model->relations, rs_len);

	for (size_t i = 0; i < rs_size; i++)
	{
		r_size = model->relations[i].size;
		r_maxsize = (r_size>r_maxsize) ? r_size : r_maxsize;
		r_len = r_size*sizeof(bituple);
		hipMalloc((void**)&(h_rs[i].tuples), r_len);
		hipMemcpy(h_rs[i].tuples, model->relations[i].tuples, r_len, hipMemcpyHostToDevice);
	}
	hipMalloc((void **)&d_rs, rs_len);
	hipMemcpy(d_rs, h_rs, rs_len, hipMemcpyHostToDevice);
	//ShowRelations << <rs_size, r_maxsize >> >(d_rs);
#pragma endregion

#pragma region ����Լ��
	int cs_size = model->cs_size;
	int cs_len = cs_size*sizeof(XMLConstraint);
	hipMalloc((void **)&d_cs, cs_len);
	hipMemcpy(d_cs, model->constraints, cs_len, hipMemcpyHostToDevice);
	//ShowConstraints << <1, cs_size >> >(d_cs);
#pragma endregion

#pragma region ����ģ��
	//hipMemcpyToSymbol(HIP_SYMBOL(d_ds), &ds_size, sizeof(int));
	//hipMemcpyToSymbol(HIP_SYMBOL(d_vs), &vs_size, sizeof(int));
	//	hipMemcpyToSymbol
	//	hipMemcpyToSymbol
	//hipMemcpyToSymbol(HIP_SYMBOL(&d_csize), &model->cs_size, sizeof(int));
	//ShowDeviceVariables << <1, 1 >> >(1);
	//XMLModel *h_model = new XMLModel;
	//XMLModel *d_model;
	//memcpy(h_model, model, sizeof(XMLModel));
	//hipMalloc((void **)&d_model, sizeof(XMLModel));
	//hipMalloc((void**)&h_model->domains, sizeof(h_dms));
	//hipMalloc((void **)&h_model->variables, sizeof(d_vs));
	//hipMalloc((void**)&h_model->variables, sizeof(h_rs));
	//hipMalloc((void **)h_model->constraints, sizeof(d_cs));
	//hipMemcpy(h_model->domains, h_dms, ds_len, hipMemcpyHostToDevice);
	//hipMemcpy(h_model->variables, model->variables, vs_len, hipMemcpyHostToDevice);
	//hipMemcpy(h_model->relations, h_rs, rs_len, hipMemcpyHostToDevice);
	//hipMemcpy(h_model->constraints, model->constraints, cs_len, hipMemcpyHostToDevice);
	//hipMemcpy(d_model, h_model, sizeof(XMLModel), hipMemcpyHostToDevice);
	//ShowModel << <1, 1 >> >(d_model);
#pragma endregion
	return true;
}
extern "C" bool BuildArcsModel(XMLModel *model)
{
	int ds_size = model->ds_size;
	int rs_size = model->rs_size;
	int cs_size = model->cs_size;
	int vs_size = model->vs_size;
#pragma region ���㻡�ֲ�/ȫ��ƫ����
	thrust::device_vector<int> d_local_counter(cs_size);
	thrust::device_vector<int> d_global_counter(cs_size, 0);
	int *d_local_counter_ptr = thrust::raw_pointer_cast(d_local_counter.data());

	thrust::device_vector<int> d_offset(cs_size, 0);
	thrust::device_vector<int> d_global_offset(cs_size, 0);
	thrust::device_vector<int> d_offset_index(cs_size);
	thrust::sequence(d_offset_index.begin(), d_offset_index.end());

	int* d_offset_ptr = thrust::raw_pointer_cast(d_offset.data());
	ComputeLocalOffset << <1, cs_size >> >(d_offset_ptr, d_local_counter_ptr, d_cs, d_dms, d_vs);

	int sum = thrust::reduce(d_offset.begin(), d_offset.end(), (int)0, thrust::plus<int>());
	counter_sum = thrust::reduce(d_local_counter.begin(), d_local_counter.end());
	d_arcs2.resize(counter_sum);

	//printf("counter_sum = %3d\n", counter_sum);

	dbsum = sum;
	//printf("edge = %d\n", dbsum / 2);
	int *d_global_offset_ptr = thrust::raw_pointer_cast(d_global_offset.data());
	int *d_global_counter_ptr = thrust::raw_pointer_cast(d_global_counter.data());
	thrust::exclusive_scan(d_offset.begin(), d_offset.end(), d_global_offset.begin());
	thrust::exclusive_scan(d_local_counter.begin(), d_local_counter.end(), d_global_counter.begin());

	d_arcs(dbsum);
#pragma endregion

#pragma region ����var_size����
	d_vars_size.resize(vs_size);
	int* d_vars_size_ptr = thrust::raw_pointer_cast(d_vars_size.data());
	//var_size *d_vars_size_ptr = thrust::raw_pointer_cast(d_vars_size.data());
	GenerateVars_size << <1, vs_size >> >(d_vars_size_ptr, d_vs, d_dms);
	//h_vars_size = d_vars_size;
#pragma endregion

#pragma region ����arcs����
	int *d_vars0_ptr = thrust::raw_pointer_cast(d_arcs.d_vars0.data());
	int *d_vals0_ptr = thrust::raw_pointer_cast(d_arcs.d_vals0.data());
	int *d_vars1_ptr = thrust::raw_pointer_cast(d_arcs.d_vars1.data());
	int *d_vals1_ptr = thrust::raw_pointer_cast(d_arcs.d_vals1.data());
	int *d_sorcs_ptr = thrust::raw_pointer_cast(d_arcs.d_sorcs.data());
	int *d_cmaps_ptr = thrust::raw_pointer_cast(d_arcs.d_cmaps.data());

	BuildArcsLaunch << <1, cs_size >> >(
		d_vars0_ptr,
		d_vals0_ptr,
		d_vars1_ptr,
		d_vals1_ptr,
		d_sorcs_ptr,
		d_cmaps_ptr,
		d_offset_ptr,
		d_global_offset_ptr,
		d_global_counter_ptr,
		d_dms,
		d_vs,
		d_rs,
		d_cs
		);

	/*int startindex = 0;
	int endindex = 0;
	h_arcs = d_arcs;

	std::cout << "input index range:" << std::endl;
	scanf("%d %d", &startindex, &endindex);
	while (!((startindex == -1) && (endindex == -1)))
	{
	if ((startindex == 0) && (endindex == 0))
	{
	for (size_t i = 0; i < dbsum; ++i)
	{
	printf("%4d:(%d,%d)--(%d,%d)=%d~%d\n",
	i,
	h_arcs.h_vars0[i],
	h_arcs.h_vals0[i],
	h_arcs.h_vars1[i],
	h_arcs.h_vals1[i],
	h_arcs.h_sorcs[i],
	h_arcs.h_cmaps[i]
	);
	}
	}
	for (size_t i = startindex; i < endindex; ++i)
	{
	printf("%4d:(%d,%d)--(%d,%d)=%d~%d\n",
	i,
	h_arcs.h_vars0[i],
	h_arcs.h_vals0[i],
	h_arcs.h_vars1[i],
	h_arcs.h_vals1[i],
	h_arcs.h_sorcs[i],
	h_arcs.h_cmaps[i]
	);
	}
	std::cout << "input index range:" << std::endl;
	scanf("%d %d", &startindex, &endindex);
	}*/
#pragma endregion

#pragma region �����ֲ�/ȫ��ƫ����
	nodes_sum = thrust::reduce(d_vars_size.begin(), d_vars_size.end());
	//printf("node = %d\n", nodes_sum);
	d_node_global.resize(vs_size);
	thrust::exclusive_scan(d_vars_size.begin(), d_vars_size.end(), d_node_global.begin());
	d_nodes_set.resize(nodes_sum, 1);
	d_nodes.resize(nodes_sum);
	int *var_ptr = thrust::raw_pointer_cast(d_nodes.vars.data());
	int *val_ptr = thrust::raw_pointer_cast(d_nodes.vals.data());
	int *nodes_offset = thrust::raw_pointer_cast(d_node_global.data());
	GenerateNodesLaunch << <1, vs_size >> >(var_ptr, val_ptr, nodes_offset, d_vs, d_dms);
#pragma endregion

#pragma region �ͷ��Դ�/�ڴ����
	for (size_t i = 0; i < ds_size; i++)
	{
		hipFree(h_dms[i].values);
	}

	hipFree(d_dms);
	delete[]h_dms;
	h_dms = NULL;

	hipFree(d_vs);

	for (size_t i = 0; i < rs_size; i++)
	{
		hipFree(h_rs[i].tuples);
	}
	hipFree(d_rs);
	delete[] h_rs;
	h_rs = NULL;
#pragma endregion

	return true;
}

extern "C" int AC4GpuPlusInitialization()
{
#pragma region ��ʼ�����ݽṹ
	int i;
	int *d_vars_key_ptr;
	int *d_vars_size_ptr;
	int vs_size = d_vars_size.size();
	int *nodes = thrust::raw_pointer_cast(d_nodes_set.data());
	int *nodes_offset = thrust::raw_pointer_cast(d_node_global.data());
	int *counter_value = thrust::raw_pointer_cast(d_arcs2.d_sorcs.data());
	hipStream_t cs[CSCOUNT];
	thrust::device_vector<int> d_vars_key(vs_size);
	D_ArcSorcIter new_end;
	d_vars_size_ptr = thrust::raw_pointer_cast(d_vars_size.data());
	d_vars_key_ptr = thrust::raw_pointer_cast(d_vars_key.data());

	for (i = 0; i < CSCOUNT; ++i)
	{
		hipStreamCreate(&(cs[i]));
	}

#pragma endregion

	thrust::reduce_by_key(
		thrust::cuda::par.on(cs[0]),
		thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.begin(), d_arcs.d_vals0.begin(), d_arcs.d_vars1.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.end(), d_arcs.d_vals0.end(), d_arcs.d_vars1.end())),
		d_arcs.d_sorcs.begin(),
		thrust::make_zip_iterator(thrust::make_tuple(d_arcs2.d_vars0.begin(), d_arcs2.d_vals0.begin(), d_arcs2.d_vars1.begin())),
		d_arcs2.d_sorcs.begin()
		);

	thrust::for_each(
		thrust::cuda::par.on(cs[0]),
		thrust::make_zip_iterator(thrust::make_tuple(d_arcs2.d_vars0.begin(), d_arcs2.d_vals0.begin(), d_arcs2.d_sorcs.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(d_arcs2.d_vars0.end(), d_arcs2.d_vals0.end(), d_arcs2.d_sorcs.end())),
		ModifyNodes(nodes, nodes_offset)
		);

	thrust::reduce_by_key(
		thrust::cuda::par.on(cs[0]),
		d_nodes.vars.begin(),
		d_nodes.vars.end(),
		d_nodes_set.begin(),
		d_vars_key.begin(),
		d_vars_size.begin()
		);

	//new_end = thrust::remove_if(
	//	thrust::cuda::par.on(cs[0]),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.begin(), d_arcs.d_vals0.begin(), d_arcs.d_vars1.begin(), d_arcs.d_vals1.begin(), d_arcs.d_sorcs.begin(), d_arcs.d_cmaps.begin())),
	//	thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.end(), d_arcs.d_vals0.end(), d_arcs.d_vars1.end(), d_arcs.d_vals1.end(), d_arcs.d_sorcs.end(), d_arcs.d_cmaps.end())),
	//	is_conflict()
	//	);
	//printf("old_end = %d", d_arcs.d_vars0.size());
	//int new_end_length = d_arcs.d_vars0.end() - thrust::get<0>(new_end.get_iterator_tuple());
	//d_arcs.resize(new_end_length);
	//printf("new_end = %d", new_end_length);

	hipStreamSynchronize(cs[0]);
	hipStreamSynchronize(cs[1]);
	hipStreamSynchronize(cs[2]);
	auto counter_has_zero = thrust::find(thrust::cuda::par.on(cs[2]), d_arcs2.d_sorcs.begin(), d_arcs2.d_sorcs.end(), 0);

	if (counter_has_zero == d_arcs2.d_sorcs.end())
	{
		std::cout << "ISAT" << std::endl;
		propagationEnable = false;
		return true;
	}

	auto domX_has_zero = thrust::find(thrust::cuda::par.on(cs[1]), d_vars_size.begin(), d_vars_size.end(), 0);

	if (domX_has_zero != d_vars_size.end())
	{
		std::cout << "IUNSAT!!" << std::endl;
		propagationEnable = false;
		return false;
	}

	hipStreamSynchronize(cs[1]);
	hipStreamSynchronize(cs[2]);

#pragma region �ͷŶ�
	for (i = 0; i < CSCOUNT; ++i)
	{
		hipStreamDestroy(cs[i]);
	}
#pragma endregion

	return 1;
}

extern "C" int AC4GpuPropagation()
{
	if (!propagationEnable)
	{
		return 0;
	}

	int i;
	int *d_vars_key_ptr;
	int *d_vars_size_ptr;
	int vs_size = d_vars_size.size();
	int *nodes = thrust::raw_pointer_cast(d_nodes_set.data());
	int *nodes_offset = thrust::raw_pointer_cast(d_node_global.data());
	int *counter_value = thrust::raw_pointer_cast(d_arcs2.d_sorcs.data());
	thrust::device_vector<int> d_vars_key(vs_size);
	thrust::device_vector<int>::iterator domX_has_zero, counter_has_zero;
	d_vars_size_ptr = thrust::raw_pointer_cast(d_vars_size.data());
	d_vars_key_ptr = thrust::raw_pointer_cast(d_vars_key.data());


	while (true)
	{
		//L7
		thrust::for_each(
			thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.begin(), d_arcs.d_vals0.begin(), d_arcs.d_vars1.begin(), d_arcs.d_vals1.begin(), d_arcs.d_sorcs.begin(), d_arcs.d_cmaps.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(d_arcs.d_vars0.end(), d_arcs.d_vals0.end(), d_arcs.d_vars1.end(), d_arcs.d_vals1.end(), d_arcs.d_sorcs.end(), d_arcs.d_cmaps.end())),
			ModifyArcs(nodes, nodes_offset, counter_value)
			);

		counter_has_zero = thrust::find(d_arcs2.d_sorcs.begin(), d_arcs2.d_sorcs.end(), (int)-1);

		if (counter_has_zero == d_arcs2.d_sorcs.end())
		{
			std::cout << "SAT" << std::endl;
			return true;
		}

		//L13
		thrust::for_each(
			thrust::make_zip_iterator(thrust::make_tuple(d_arcs2.d_vars0.begin(), d_arcs2.d_vals0.begin(), d_arcs2.d_sorcs.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(d_arcs2.d_vars0.end(), d_arcs2.d_vals0.end(), d_arcs2.d_sorcs.end())),
			ModifyNodesProp(nodes, nodes_offset, d_vars_size_ptr)
			);

		domX_has_zero = thrust::find(d_vars_size.begin(), d_vars_size.end(), (int)0);

		if (domX_has_zero != d_vars_size.end())
		{

			thrust::reduce_by_key(
				d_nodes.vars.begin(),
				d_nodes.vars.end(),
				d_nodes_set.begin(),
				d_vars_key.begin(),
				d_vars_size.begin()
				);

			domX_has_zero = thrust::find(d_vars_size.begin(), d_vars_size.end(), (int)0);

			if (domX_has_zero != d_vars_size.end())
			{

				std::cout << "UNSAT!!" << std::endl;
				return false;
			}
		}
	}
	return 1;
};